#include "hip/hip_runtime.h"
/*
* Copyright (C) 2022 Ricardo J. Barrientos (rbarrientos@ucm.cl)
*
* This program is free software: you can redistribute it and/or modify
* it under the terms of the GNU General Public License as published by
* the Free Software Foundation, either version 3 of the License, or
* (at your option) any later version.
*
* This program is distributed in the hope that it will be useful,
* but WITHOUT ANY WARRANTY; without even the implied warranty of
* MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
* GNU General Public License for more details.
*
* You should have received a copy of the GNU General Public License
* along with this program.  If not, see <https://www.gnu.org/licenses/>.
*/

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <sys/resource.h>
#include <time.h>
#include <sys/time.h>

//Dimension of the elements
#define DIM 254

//The value of the K elements to be retrieved
#define TOPK 32

//Number of elements of the database
#define NE 1000000

//Number of threads of a warp in GPU
#define TAM_WARP 32

//Number of threads of a CUDA Block
#define TAM_MAX_BLOCK 352

//Q is the quantity of queries in each batch. It is limited by the memory
#define Q 3972 //with 95325 elements (vectors of dimension 254)
//#define Q 2979 //with 200000 elements (vectors of dimension 254)
//#define Q 1324 //with 500000 elements (vectors of dimension 254)
//#define Q 662 //with 999996 elements (vectors of dimension 254)
//#define Q 442 //with 1500000 elements (vectors of dimension 254)
//#define Q 331 //with 2000000 elements (vectors of dimension 254)


//Structure to keep the distance (to and from the query) and the index of the result
struct _Elem
{
  float dist;
  int ind;
};
typedef struct _Elem Elem;


//functions
__device__ void insertH(Elem *heap, Elem *elem, int *n_elem, int pitch, int id);
__device__ void extractH(Elem *heap, int *n_elem, int pitch, int id, Elem *eresult);
__device__ float topH(Elem *heap, int id);
__device__ void popush(Elem *heap, Elem *elem, int *n_elem, int pitch, int id);
__global__ void Batch_Heap_Reduction(float *DB_dev, int pitch_DB, Elem *heap, int pitch_H, float *QUERY_dev, int pitch_QUERY, Elem *arr_Dist, int pitch_Dist, int beginQ, float *res_final);
__device__ float distancia_trans(float *p1, int pitch_p1, int col_1, float *q);
int readdata(float *dato, FILE *file);


int N_QUERIES;


main(int argc, char *argv[])
{
   int i, N_ELEM, dimension, tam_elem, j;
   FILE *pf;
   float **vectores;
   float  real_time;
   struct timeval t1, t2;
   float *Elems, *QUERY_dev;
   float **consultas, *res_final, *res_final_H;
   int retorno, T_per_BLOCK, N_BLOQUES, contQ, cont;
   Elem *HEAPS_dev, *arr_res1, *arr_res1H, *arr_Dist;
   size_t pitch, pitch_H, pitch_Q, pitch_Dist;
   int *resT, *resTH;
   long long ED_total=0;
   double prom, prom_cont;


   if (argc != 6)
   {
      printf("\nExecute as: ./a.out file_BD file_queries DB_Size Queries_Size DIMENSION\n");
      return 0;
   }
   if ((pf = fopen(argv[1], "r")) == NULL)
   {
      printf("\nCannot open the file %s\n" ,argv[1]);
      return 0;
   }

   N_ELEM = atoi(argv[3]);
   N_QUERIES = atoi(argv[4]);
   dimension = atoi(argv[5]);

   if (dimension != DIM )
   {
     printf("\nERROR :: dimension != DIM\n");
     return 0;
   }

   printf("\nCant. Elementos=%d :: dimension=%d\n" , N_ELEM, dimension);
   fflush(stdout);
   if (N_ELEM != NE)
   {
     printf("\nERORR :: N_ELEM != NE\n");
     return 0;
   }
   if (N_ELEM < 512*32)
   {
     printf("\nERROR :: No enough database size to keep TOPK elements in each heap\n");
     return 0;
   }

   if (hipSuccess != hipMalloc((void **)&res_final, sizeof(float)*Q))
   {
     printf("\nERROR 21 :: hipMalloc\n");
     hipDeviceReset();
     return 0;
   }
   res_final_H = (float *)malloc(sizeof(float)*Q);
   for (i=0; i<Q; i++)
   {
      res_final_H[i] = 0;
   }
   if (hipSuccess != hipMemset(res_final, 0, sizeof(float)*Q))
   {
       printf("\nERROR :: hipMemset\n");
       hipDeviceReset();
       return 0;
   }

   //HEAPS_dev[TOPK][Q*512]
   if (hipSuccess != hipMallocPitch((void **)&HEAPS_dev, &pitch_H, sizeof(Elem)*Q*512, (size_t)TOPK))
   {
      printf("\nERROR 4 :: hipMallocPitch :: Heaps_dev col=%lld :: row=%d\n", (long long)(sizeof(Elem)*Q*512), TOPK);
      hipDeviceReset();
      return 0;
   }

   //arr_Dist[Q][N_ELEM]
   if (hipSuccess != hipMallocPitch((void **)&arr_Dist, &pitch_Dist, N_ELEM*sizeof(Elem), (size_t)Q))
   {
      printf("\nERROR 41 :: hipMallocPitch\n");
      hipDeviceReset();
      return 0;
   }

   vectores =(float **)malloc(sizeof(float *)*dimension);
   for (i=0; i<dimension; i++)
      vectores[i] = (float *)malloc(sizeof(float)*N_ELEM);

   //Reading the elements of the database
   for (i=0; i<N_ELEM; i++)
   {
//      printf("Reading vectors[%d] : ", i);
      for (j=0; j<dimension; j++)
      {
         fscanf(pf, "%f", &vectores[j][i]);
//         printf("%f ", vectores[i][j]);
      }
//      printf("\n");
      fgetc(pf);
   }
   fclose(pf);

   //Elems[dimension][N_ELEM]
   if (hipSuccess != hipMallocPitch((void **)&Elems, (size_t *)&pitch, N_ELEM*sizeof(float), (size_t)dimension))
      printf("\nERROR :: hipMallocPitch 4\n");

   for (i=0; i < dimension; i++)
   {
      retorno = hipMemcpy((float *)((char *)Elems + (i*(int)pitch)), (float *)(vectores[i]), sizeof(float)*N_ELEM, hipMemcpyHostToDevice);
     if (retorno != hipSuccess)
     {
      switch(retorno)
      {
       case hipErrorInvalidPitchValue:
         printf("\nERROR 2 -> hipErrorInvalidPitchValue:\n");
         break;
       case hipErrorInvalidDevicePointer:
         printf("\nERROR 2 -> hipErrorInvalidDevicePointer:\n");
         break;
       case hipErrorInvalidMemcpyDirection:
         printf("\nERROR 2 -> hipErrorInvalidMemcpyDirection:\n");
         break;
       case hipErrorInvalidValue:
         printf("\nERROR 2 -> hipErrorInvalidValue :: i=%d :: pitch=%d\n", i, pitch);
         break;
       default: 
         printf("\nERROR 2 -> Checkear esto.\n");
         break;
      }
      return 0;
     }
   }

   consultas =(float **)malloc(sizeof(float *)*N_QUERIES);
   for (i=0; i<N_QUERIES; i++)
      consultas[i] = (float *)malloc(sizeof(float)*dimension);

   if ((pf = fopen(argv[2], "r")) == NULL)
   {
      printf("\nNo se pudo abrir el archivo %s\n" ,argv[2]);
      return 0;
   }
/*
   fgets(linea, tam_lin-1, pf);
   fscanf(pf, "%d", &N_QUERIES);
   fscanf(pf, "%d", &dimension);
   fscanf(pf, "%d", &tam_elem);
   fgetc(pf);
   */
   printf("\n\nArchivo de Queries:\nCant. Elementos=%d :: dimension=%d\n" , N_QUERIES, dimension);

   //Reading the queries
   for (i=0; i<N_QUERIES; i++)
   {
	if (readdata(consultas[i], pf) == -1)
	{
		printf("\nError al leer Consultas\n");
         	hipDeviceReset();
		return 0;
	}
   }
   fclose(pf);

   //QUERY_dev[N_QUERIES][dimension]
   if (hipSuccess != hipMallocPitch((void **)&QUERY_dev, (size_t *)&pitch_Q, dimension*sizeof(float), (size_t)N_QUERIES))
      printf("\nERROR :: hipMallocPitch 1\n");

   for (i=0; i < N_QUERIES; i++)
   {
     if (hipSuccess != hipMemcpy((char *)QUERY_dev + (i*(int)pitch_Q), consultas[i], sizeof(float)*dimension, hipMemcpyHostToDevice))
       printf("\nERROR 3 :: hipMemcpy\n");
   }

   //----------------------------
   T_per_BLOCK = N_ELEM;
   if (T_per_BLOCK > 512)
      T_per_BLOCK = 512;
   

   N_BLOQUES = Q;
   contQ = 0;
   cont = 0;
   getrusage(RUSAGE_SELF, &r1);
   gettimeofday(&t1, 0);

  while(contQ < N_QUERIES)
  {

      contQ += Q;
      if (contQ > N_QUERIES)
         N_BLOQUES = N_QUERIES - (contQ-Q);
      printf("\nN_BLOQUES = %d :: T_per_BLOCK = %d\n", N_BLOQUES, T_per_BLOCK);

      Batch_Heap_Reduction<<< N_BLOQUES, T_per_BLOCK>>> (Elems, (int)pitch, HEAPS_dev, (int)pitch_H, QUERY_dev, (int)pitch_Q, arr_Dist, (int)pitch_Dist, Q*cont, res_final);

      if (hipSuccess != hipMemcpy((float *)res_final_H, (float *)res_final, sizeof(float)*Q, hipMemcpyDeviceToHost))
      {
         printf("\nERROR 41 :: hipMemcpy :: iteraH\n");
         hipDeviceReset();
         return 0;
      }
      cont++;
  }

  gettimeofday(&t2, 0);


   real_time = (t2.tv_sec - t1.tv_sec) + (float)(t2.tv_usec - t1.tv_usec)/1000000;

   prom = 0;
   prom_cont = 0;
   for (i=0; i<Q; i++)
   {
         if (res_final_H[i] != 0)
         {
            prom += res_final_H[i];
            prom_cont += 1;
         }
   }

   printf("\nK = %d", TOPK);
   printf("\nReal Time = %f", real_time);
   printf("\nprom = %lf\n", (double)(prom/(double)prom_cont));
   fflush(stdout);

   hipFree(Elems);
   hipFree(QUERY_dev);
   hipFree(HEAPS_dev);
   hipFree(arr_Dist);

  hipDeviceReset();
  return 0;
}


__device__ void insertH(Elem *heap, Elem *elem, int *n_elem, int pitch, int id)
{
  int i;
  Elem temp;

  ((Elem *)((char *)heap + (*n_elem)*pitch))[id].dist = elem->dist;
  ((Elem *)((char *)heap + (*n_elem)*pitch))[id].ind = elem->ind;
  (*n_elem)++;
  for (i = *n_elem; i>1 && ((Elem *)((char *)heap + (i-1)*pitch))[id].dist > ((Elem *)((char *)heap + ((i/2)-1)*pitch))[id].dist; i=i/2)
  {
    //Intercambiamos con el padre
    temp.dist = ((Elem *)((char *)heap + (i-1)*pitch))[id].dist;
    temp.ind = ((Elem *)((char *)heap + (i-1)*pitch))[id].ind;
    ((Elem *)((char *)heap + (i-1)*pitch))[id].dist = ((Elem *)((char *)heap + ((i/2)-1)*pitch))[id].dist;
    ((Elem *)((char *)heap + (i-1)*pitch))[id].ind = ((Elem *)((char *)heap + ((i/2)-1)*pitch))[id].ind;
    ((Elem *)((char *)heap + ((i/2)-1)*pitch))[id].dist = temp.dist;
    ((Elem *)((char *)heap + ((i/2)-1)*pitch))[id].ind = temp.ind;
  }
  return;
}

__device__ void extractH(Elem *heap, int *n_elem, int pitch, int id, Elem *eresult)
{
  int i, k;
  Elem temp;
  eresult->dist = ((Elem *)((char *)heap+0))[id].dist; //Se guarda el maximo
  eresult->ind = ((Elem *)((char *)heap+0))[id].ind; 

  ((Elem *)((char *)heap+0))[id].dist = ((Elem *)((char *)heap + ((*n_elem)-1)*pitch))[id].dist;// Movemos el ultimo a la raiz y achicamos el heap
  ((Elem *)((char *)heap+0))[id].ind = ((Elem *)((char *)heap + ((*n_elem)-1)*pitch))[id].ind;
  (*n_elem)--;
  i = 1;
  while(2*i <= *n_elem) // mientras tenga algun hijo
  {
    k = 2*i; //el hijo izquierdo
    if(k+1 <= *n_elem && ((Elem *)((char *)heap + ((k+1)-1)*pitch))[id].dist > ((Elem *)((char *)heap + (k-1)*pitch))[id].dist)
      k = k+1;  //el hijo derecho es el mayor

    if(((Elem *)((char *)heap + (i-1)*pitch))[id].dist > ((Elem *)((char *)heap + (k-1)*pitch))[id].dist)
      break;  //es mayor que ambos hijos

    temp.dist = ((Elem *)((char *)heap + (i-1)*pitch))[id].dist;
    temp.ind = ((Elem *)((char *)heap + (i-1)*pitch))[id].ind;
    ((Elem *)((char *)heap + (i-1)*pitch))[id].dist = ((Elem *)((char *)heap + (k-1)*pitch))[id].dist;
    ((Elem *)((char *)heap + (i-1)*pitch))[id].ind = ((Elem *)((char *)heap + (k-1)*pitch))[id].ind;
    ((Elem *)((char *)heap + (k-1)*pitch))[id].dist = temp.dist;
    ((Elem *)((char *)heap + (k-1)*pitch))[id].ind = temp.ind;
    i = k;   //lo intercambiamos con el mayor hijo
  }
  return;
//  return max;
}


__device__ float topH(Elem *heap, int id)
{
  return ((Elem *)((char *)heap + 0))[id].dist;
}

__device__ void popush(Elem *heap, Elem *elem, int *n_elem, int pitch, int id)
{
  int i, k;
  Elem temp;

  ((Elem *)((char *)heap+0))[id].dist = elem->dist;
  ((Elem *)((char *)heap+0))[id].ind  = elem->ind;

  i = 1;
  while(2*i <= *n_elem) // mientras tenga algun hijo
  {
    k = 2*i; //el hijo izquierdo
    if(k+1 <= *n_elem && ((Elem *)((char *)heap + ((k+1)-1)*pitch))[id].dist > ((Elem *)((char *)heap + (k-1)*pitch))[id].dist)
      k = k+1;  //el hijo derecho es el mayor

    if(((Elem *)((char *)heap + (i-1)*pitch))[id].dist > ((Elem *)((char *)heap + (k-1)*pitch))[id].dist)
      break;  //es mayor que ambos hijos

    temp.dist = ((Elem *)((char *)heap + (i-1)*pitch))[id].dist;
    temp.ind = ((Elem *)((char *)heap + (i-1)*pitch))[id].ind;
    ((Elem *)((char *)heap + (i-1)*pitch))[id].dist = ((Elem *)((char *)heap + (k-1)*pitch))[id].dist;
    ((Elem *)((char *)heap + (i-1)*pitch))[id].ind = ((Elem *)((char *)heap + (k-1)*pitch))[id].ind;
    ((Elem *)((char *)heap + (k-1)*pitch))[id].dist = temp.dist;
    ((Elem *)((char *)heap + (k-1)*pitch))[id].ind = temp.ind;
    i = k;   //lo intercambiamos con el mayor hijo
  }
  return;
}


__global__ void Batch_Heap_Reduction(float *DB_dev, int pitch_DB, Elem *heap, int pitch_H, float *QUERY_dev, int pitch_QUERY, Elem *arr_Dist, int pitch_Dist, int beginQ, float *res_final)
{
  int i, j, n_elem=0, n_elemWarp=0;
  int id;
  Elem eresult;
  __shared__ Elem matrizWarp[TOPK][TAM_WARP];
  __shared__ Elem heapfin[TOPK][1];
  __shared__ float query[DIM];

  id = threadIdx.x + (blockDim.x * blockIdx.x);

  //Copying the query to shared memory
  for (i=threadIdx.x; i < DIM; i += blockDim.x)
      query[i] = ((float *)((char *)QUERY_dev + ((blockIdx.x + beginQ) * (int)pitch_QUERY)))[i];

  __syncthreads();

  //Getting the array of distances
  for (i=threadIdx.x; i < NE; i += blockDim.x)
  {
    ((Elem *)((char *)arr_Dist + (blockIdx.x*pitch_Dist)))[i].dist = distancia_trans(DB_dev, pitch_DB, i, query);
    ((Elem *)((char *)arr_Dist + (blockIdx.x*pitch_Dist)))[i].ind = i;
  }

  for(i=threadIdx.x; i < NE; i += blockDim.x)//NE = Numero de elementos de la BD
  {
      if (n_elem >= TOPK)
      {
         if (topH(heap, id) > ((Elem *)((char *)arr_Dist + (blockIdx.x*pitch_Dist)))[i].dist)
            popush(heap, &(((Elem *)((char *)arr_Dist + (blockIdx.x*pitch_Dist)))[i]), &n_elem, pitch_H, id); //pop and push in a single operation
      }
      else
         insertH(heap, &(((Elem *)((char *)arr_Dist + (blockIdx.x*pitch_Dist)))[i]), &n_elem, pitch_H, id);
  }

  __syncthreads();


  //The first warp of the CUDA Block reduces the problem to a matrix of size Kx32 elements, but storing the elements of the heaps in shared memory
  if (threadIdx.x < TAM_WARP)
  {
    for(j=id; j < blockDim.x*(blockIdx.x+1); j += TAM_WARP)
    {
       n_elem = TOPK;
       for(i=0; i < TOPK; i++)
       {
         extractH(heap, &n_elem, pitch_H, j, &eresult);

         if (n_elemWarp < TOPK)
           insertH(&(matrizWarp[0][0]), &eresult, &n_elemWarp, sizeof(Elem)*TAM_WARP, threadIdx.x);
         else
           if (topH(&(matrizWarp[0][0]), threadIdx.x) > eresult.dist)
             popush(&(matrizWarp[0][0]), &eresult, &n_elemWarp, sizeof(Elem)*TAM_WARP, threadIdx.x);
       }
    }
  }
  

  __syncthreads();


  //the first thread of the CUDA Block finds the k results from the previous matrix of size TOPKxTAM_WARP
  if (threadIdx.x == 0)
  {
     n_elem = 0;
     for(j=0; j < TAM_WARP; j++)
     {
       for(i=0; i < TOPK; i++)
          if (n_elem < TOPK)
             insertH((Elem *)heapfin, &(matrizWarp[i][j]), &n_elem, sizeof(Elem), 0);
          else
             if (topH((Elem *)heapfin, 0) > matrizWarp[i][j].dist)
               popush((Elem *)heapfin, &(matrizWarp[i][j]), &n_elem, sizeof(Elem), 0);
     }

     //Writing the closest element to the query
     res_final[blockIdx.x] = topH((Elem *)heapfin, 0);

     //To write te TOPK elements retrieved in this function, you must use an array 'arr_res1' with size TOPK*Q, and to do as follows:
//     for (i=TOPK*blockIdx.x; i<TOPK*(blockIdx.x+1); i++)
//        extractH(&(heapfin[0][0]), &n_elem, sizeof(Elem), 0, &(arr_res1[i]));
  }

  return;
}

__device__ float distancia_trans(float *p1, int pitch_p1, int col_1, float *q)
{
   int i=0;
   float suma=0;

   for (i=0; i < DIM; i++)
      suma += (((float *)((char *)p1 + (i*pitch_p1)))[col_1] - q[i]) * 
              (((float *)((char *)p1 + (i*pitch_p1)))[col_1] - q[i]);

   return sqrtf(suma);
}


int readdata(float *dato, FILE *file)
{
   int i=0;
   
   for (i=0;i<DIM;i++)
      if (fscanf(file,"%f",&dato[i])<1)
         return -1;
   return 1;
}
